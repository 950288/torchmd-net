#include "hip/hip_runtime.h"
/* Raul P. Pelaez 2023. Batched cell list neighbor list implementation for CUDA.

 */
#include "common.cuh"
#include <cub/device/device_radix_sort.cuh>
#include <thrust/device_malloc_allocator.h>
#include <thrust/device_vector.h>

/*
 * @brief Encodes an unsigned integer lower than 1024 as a 32 bit integer by filling every third
 * bit.
 * @param i The integer to encode
 * @return The encoded integer
 */
inline __host__ __device__ uint encodeMorton(const uint& i) {
    uint x = i;
    x &= 0x3ff;
    x = (x | x << 16) & 0x30000ff;
    x = (x | x << 8) & 0x300f00f;
    x = (x | x << 4) & 0x30c30c3;
    x = (x | x << 2) & 0x9249249;
    return x;
}

/*
 * @brief Interleave three 10 bit numbers in 32 bits, producing a Z order Morton hash
 * @param ci The cell index
 * @return The Morton hash
 */
inline __host__ __device__ uint hashMorton(int3 ci) {
    return encodeMorton(ci.x) | (encodeMorton(ci.y) << 1) | (encodeMorton(ci.z) << 2);
}

/*
 * @brief Calculates the cell dimensions for a given box size and cutoff
 * @param box_size The box size
 * @param cutoff The cutoff
 * @return The cell dimensions
 */
template <typename scalar_t>
__host__ __device__ int3 getCellDimensions(scalar3<scalar_t> box_size, scalar_t cutoff) {
    int3 cell_dim = make_int3(box_size.x / cutoff, box_size.y / cutoff, box_size.z / cutoff);
    // Minimum 3 cells in each dimension
    cell_dim.x = thrust::max(cell_dim.x, 3);
    cell_dim.y = thrust::max(cell_dim.y, 3);
    cell_dim.z = thrust::max(cell_dim.z, 3);
// In the host, throw if there are more than 1024 cells in any dimension
#ifndef __CUDA_ARCH__
    if (cell_dim.x > 1024 || cell_dim.y > 1024 || cell_dim.z > 1024) {
        throw std::runtime_error("Too many cells in one dimension. Maximum is 1024");
    }
#endif
    return cell_dim;
}

/*
 * @brief Get the cell index of a point
 * @param p The point position
 * @param box_size The size of the box in each dimension
 * @param cutoff The cutoff
 * @return The cell index
 */
template <typename scalar_t>
__device__ int3 getCell(scalar3<scalar_t> p, scalar3<scalar_t> box_size, scalar_t cutoff) {
    p = rect::apply_pbc<scalar_t>(p, box_size);
    // Take to the [0, box_size] range and divide by cutoff (which is the cell size)
    int cx = floorf((p.x + scalar_t(0.5) * box_size.x) / cutoff);
    int cy = floorf((p.y + scalar_t(0.5) * box_size.y) / cutoff);
    int cz = floorf((p.z + scalar_t(0.5) * box_size.z) / cutoff);
    int3 cell_dim = getCellDimensions(box_size, cutoff);
    // Wrap around. If the position of a particle is exactly box_size, it will be in the last cell,
    // which results in an illegal access down the line.
    if (cx == cell_dim.x)
        cx = 0;
    if (cy == cell_dim.y)
        cy = 0;
    if (cz == cell_dim.z)
        cz = 0;
    return make_int3(cx, cy, cz);
}

/*
 * @brief Get the index of a cell in a 1D array of cells.
 * @param cell The cell coordinates, assumed to be in the range [0, cell_dim].
 * @param cell_dim The number of cells in each dimension
 */
__device__ int getCellIndex(int3 cell, int3 cell_dim) {
    return cell.x + cell_dim.x * (cell.y + cell_dim.y * cell.z);
}

/*
  @brief Fold a cell coordinate to the range [0, cell_dim)
  @param cell The cell coordinate
  @param cell_dim The dimensions of the grid
  @return The folded cell coordinate
*/
__device__ int3 getPeriodicCell(int3 cell, int3 cell_dim) {
    int3 periodic_cell = cell;
    if (cell.x < 0)
        periodic_cell.x += cell_dim.x;
    if (cell.x >= cell_dim.x)
        periodic_cell.x -= cell_dim.x;
    if (cell.y < 0)
        periodic_cell.y += cell_dim.y;
    if (cell.y >= cell_dim.y)
        periodic_cell.y -= cell_dim.y;
    if (cell.z < 0)
        periodic_cell.z += cell_dim.z;
    if (cell.z >= cell_dim.z)
        periodic_cell.z -= cell_dim.z;
    return periodic_cell;
}

// Assign a hash to each atom based on its position and batch.
// This hash is such that atoms in the same cell and batch have the same hash.
template <typename scalar_t>
__global__ void assignHash(const Accessor<scalar_t, 2> positions, uint64_t* hash_keys,
                           Accessor<int32_t, 1> hash_values, const Accessor<int64_t, 1> batch,
                           scalar3<scalar_t> box_size, scalar_t cutoff, int32_t num_atoms) {
    const int32_t i_atom = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_atom >= num_atoms)
        return;
    const int32_t i_batch = batch[i_atom];
    // Move to the unit cell
    scalar3<scalar_t> pi = {positions[i_atom][0], positions[i_atom][1], positions[i_atom][2]};
    auto ci = getCell(pi, box_size, cutoff);
    // Calculate the hash
    const int32_t hash = hashMorton(ci);
    // Create a hash combining the Morton hash and the batch index, so that atoms in the same cell
    // are contiguous
    const int64_t hash_final = (static_cast<int64_t>(hash) << 32) | i_batch;
    hash_keys[i_atom] = hash_final;
    hash_values[i_atom] = i_atom;
}

/*
 * @brief A buffer that is allocated and deallocated using the CUDA caching allocator from torch
 */
template <class T> struct cached_buffer {
    cached_buffer(size_t size) : size_(size) {
        ptr_ = static_cast<T*>(at::cuda::CUDACachingAllocator::raw_alloc(size * sizeof(T)));
    }
    ~cached_buffer() {
        at::cuda::CUDACachingAllocator::raw_delete(ptr_);
    }
    T* get() {
        return ptr_;
    }
    size_t size() {
        return size_;
    }

private:
    T* ptr_;
    size_t size_;
};

/*
 * @brief Sort the positions by hash, first by the cell assigned to each position and the batch
 * index
 * @param positions The positions of the atoms
 * @param batch The batch index of each atom
 * @param box_size The box vectors
 * @param cutoff The cutoff
 * @return A tuple of the sorted positions and the original indices of each atom in the sorted list
 */
static auto sortPositionsByHash(const Tensor& positions, const Tensor& batch,
                                const Tensor& box_size, const Scalar& cutoff) {
    const int num_atoms = positions.size(0);
    auto hash_keys = cached_buffer<uint64_t>(num_atoms);
    Tensor hash_values = empty({num_atoms}, positions.options().dtype(torch::kInt32));
    const int threads = 128;
    const int blocks = (num_atoms + threads - 1) / threads;
    auto stream = at::cuda::getCurrentCUDAStream();
    AT_DISPATCH_FLOATING_TYPES(positions.scalar_type(), "assignHash", [&] {
        scalar_t cutoff_ = cutoff.to<scalar_t>();
        scalar3<scalar_t> box_size_ = {box_size[0][0].item<scalar_t>(),
                                       box_size[1][1].item<scalar_t>(),
                                       box_size[2][2].item<scalar_t>()};
        assignHash<<<blocks, threads, 0, stream>>>(
            get_accessor<scalar_t, 2>(positions), hash_keys.get(),
            get_accessor<int32_t, 1>(hash_values), get_accessor<int64_t, 1>(batch), box_size_,
            cutoff_, num_atoms);
    });
    // I have to use cub directly because thrust::sort_by_key is not compatible with graphs
    //  and torch::lexsort does not support uint64_t
    size_t tmp_storage_bytes = 0;
    auto d_keys_out = cached_buffer<uint64_t>(num_atoms);
    auto d_values_out = cached_buffer<int32_t>(num_atoms);
    int32_t* hash_values_ptr = hash_values.data_ptr<int32_t>();
    hipcub::DeviceRadixSort::SortPairs(nullptr, tmp_storage_bytes, hash_keys.get(), d_keys_out.get(),
                                    hash_values_ptr, d_values_out.get(), num_atoms, 0, 64, stream);
    auto tmp_storage = cached_buffer<char>(tmp_storage_bytes);
    hipcub::DeviceRadixSort::SortPairs(tmp_storage.get(), tmp_storage_bytes, hash_keys.get(),
                                    d_keys_out.get(), hash_values_ptr, d_values_out.get(),
                                    num_atoms, 0, 64, stream);
    hipMemcpyAsync(hash_values_ptr, d_values_out.get(), num_atoms * sizeof(int32_t),
                    hipMemcpyDeviceToDevice, stream);
    Tensor sorted_positions = positions.index_select(0, hash_values);
    return std::make_tuple(sorted_positions, hash_values);
}

template <typename scalar_t>
__global__ void fillCellOffsetsD(const Accessor<scalar_t, 2> sorted_positions,
                                 const Accessor<int32_t, 1> sorted_indices,
                                 Accessor<int32_t, 1> cell_start, Accessor<int32_t, 1> cell_end,
                                 const Accessor<int64_t, 1> batch, scalar3<scalar_t> box_size,
                                 scalar_t cutoff) {
    // Since positions are sorted by cell, for a given atom, if the previous atom is in a different
    // cell, then the current atom is the first atom in its cell We use this fact to fill the
    // cell_start and cell_end arrays
    const int32_t i_atom = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_atom >= sorted_positions.size(0))
        return;
    const scalar3<scalar_t> pi = {sorted_positions[i_atom][0], sorted_positions[i_atom][1],
                                  sorted_positions[i_atom][2]};
    const int3 cell_dim = getCellDimensions(box_size, cutoff);
    const int icell = getCellIndex(getCell(pi, box_size, cutoff), cell_dim);
    int im1_cell;
    if (i_atom > 0) {
        int im1 = i_atom - 1;
        const scalar3<scalar_t> pim1 = {sorted_positions[im1][0], sorted_positions[im1][1],
                                        sorted_positions[im1][2]};
        im1_cell = getCellIndex(getCell(pim1, box_size, cutoff), cell_dim);
    } else {
        im1_cell = 0;
    }
    if (icell != im1_cell || i_atom == 0) {
        int n_cells = cell_start.size(0);
        cell_start[icell] = i_atom;
        if (i_atom > 0)
            cell_end[im1_cell] = i_atom;
    }
    if (i_atom == sorted_positions.size(0) - 1) {
        cell_end[icell] = i_atom + 1;
    }
}

/*
  @brief
  Fill the cell offsets for each batch, identifying the start and end of each cell in the sorted
  positions
  @param sorted_positions The positions sorted by cell
  @param sorted_indices The original indices of the sorted positions
  @param batch The batch index of each position
  @param box_size The box vectors
  @param cutoff The cutoff distance
  @return A tuple of cell_start and cell_end arrays
*/
static auto fillCellOffsets(const Tensor& sorted_positions, const Tensor& sorted_indices,
                            const Tensor& batch, const Tensor& box_size, const Scalar& cutoff) {
    const TensorOptions options = sorted_positions.options();
    int3 cell_dim;
    AT_DISPATCH_FLOATING_TYPES(sorted_positions.scalar_type(), "fillCellOffsets", [&] {
        scalar_t cutoff_ = cutoff.to<scalar_t>();
        scalar3<scalar_t> box_size_ = {box_size[0][0].item<scalar_t>(),
                                       box_size[1][1].item<scalar_t>(),
                                       box_size[2][2].item<scalar_t>()};
        cell_dim = getCellDimensions(box_size_, cutoff_);
    });
    const int num_cells = cell_dim.x * cell_dim.y * cell_dim.z;
    const Tensor cell_start = full({num_cells}, -1, options.dtype(torch::kInt));
    const Tensor cell_end = empty({num_cells}, options.dtype(torch::kInt));
    const int threads = 128;
    const int blocks = (sorted_positions.size(0) + threads - 1) / threads;
    AT_DISPATCH_FLOATING_TYPES(sorted_positions.scalar_type(), "fillCellOffsets", [&] {
        auto stream = at::cuda::getCurrentCUDAStream();
        scalar_t cutoff_ = cutoff.to<scalar_t>();
        scalar3<scalar_t> box_size_ = {box_size[0][0].item<scalar_t>(),
                                       box_size[1][1].item<scalar_t>(),
                                       box_size[2][2].item<scalar_t>()};
        fillCellOffsetsD<<<blocks, threads, 0, stream>>>(
            get_accessor<scalar_t, 2>(sorted_positions), get_accessor<int32_t, 1>(sorted_indices),
            get_accessor<int32_t, 1>(cell_start), get_accessor<int32_t, 1>(cell_end),
            get_accessor<int64_t, 1>(batch), box_size_, cutoff_);
    });
    return std::make_tuple(cell_start, cell_end);
}

/*
  @brief Get the cell index of the i'th neighboring cell for a given cell
  @param cell_i The cell coordinates
  @param i The index of the neighboring cell, from 0 to 26
  @param cell_dim The dimensions of the cell grid
  @return The cell index of the i'th neighboring cell
*/
__device__ int getNeighborCellIndex(int3 cell_i, int i, int3 cell_dim) {
    auto cell_j = cell_i;
    cell_j.x += i % 3 - 1;
    cell_j.y += (i / 3) % 3 - 1;
    cell_j.z += i / 9 - 1;
    cell_j = getPeriodicCell(cell_j, cell_dim);
    int icellj = getCellIndex(cell_j, cell_dim);
    return icellj;
}

// Traverse the cell list for each atom and find the neighbors
template <typename scalar_t>
__global__ void
forward_kernel(const Accessor<scalar_t, 2> sorted_positions,
               const Accessor<int32_t, 1> original_index, const Accessor<int64_t, 1> batch,
               const Accessor<int32_t, 1> cell_start, const Accessor<int32_t, 1> cell_end,
               Accessor<int32_t, 2> neighbors, Accessor<scalar_t, 2> deltas,
               Accessor<scalar_t, 1> distances, Accessor<int32_t, 1> i_curr_pair, int num_atoms,
               int num_pairs, bool use_periodic, scalar3<scalar_t> box_size, scalar_t cutoff_lower,
               scalar_t cutoff_upper, bool loop, bool include_transpose) {
    // Each atom traverses the cells around it and finds the neighbors
    // Atoms for all batches are placed in the same cell list, but other batches are ignored while
    // traversing
    const int32_t i_atom = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_atom >= num_atoms)
        return;
    const int ori = original_index[i_atom];
    const auto i_batch = batch[ori];
    const scalar3<scalar_t> pi = {sorted_positions[i_atom][0], sorted_positions[i_atom][1],
                                  sorted_positions[i_atom][2]};
    const int3 cell_i = getCell(pi, box_size, cutoff_upper);
    const int3 cell_dim = getCellDimensions(box_size, cutoff_upper);
    // Loop over the 27 cells around the current cell
    for (int i = 0; i < 27; i++) {
        int icellj = getNeighborCellIndex(cell_i, i, cell_dim);
        const int firstParticle = cell_start[icellj];
        if (firstParticle != -1) { // Continue only if there are particles in this cell
            // Index of the last particle in the cell's list
            const int lastParticle = cell_end[icellj];
            const int nincell = lastParticle - firstParticle;
            for (int j = 0; j < nincell; j++) {
                const int cur_j = j + firstParticle;
                const int orj = original_index[cur_j];
                const auto j_batch = batch[orj];
                if (j_batch >
                    i_batch) // Particles are sorted by batch after cell, so we can break early here
                    break;
                const bool testPair =
                    (j_batch == i_batch) and ((orj < ori) or (loop and orj == ori));
                if (testPair) {
                    const scalar3<scalar_t> pj = {sorted_positions[cur_j][0],
                                                  sorted_positions[cur_j][1],
                                                  sorted_positions[cur_j][2]};
                    const auto delta =
                        rect::compute_distance<scalar_t>(pi, pj, use_periodic, box_size);
                    const scalar_t distance2 =
                        delta.x * delta.x + delta.y * delta.y + delta.z * delta.z;
                    const scalar_t cutoff_upper2 = cutoff_upper * cutoff_upper;
                    const scalar_t cutoff_lower2 = cutoff_lower * cutoff_lower;
                    if ((distance2 < cutoff_upper2 and distance2 >= cutoff_lower2) or
                        (loop and orj == ori)) {
                        const bool requires_transpose = include_transpose and (orj != ori);
                        const int32_t i_pair =
                            atomicAdd(&i_curr_pair[0], requires_transpose ? 2 : 1);
                        // We handle too many neighbors outside of the kernel
                        if (i_pair + requires_transpose < neighbors.size(1)) {
                            const scalar_t distance = sqrt_(distance2);
                            neighbors[0][i_pair] = ori;
                            neighbors[1][i_pair] = orj;
                            deltas[i_pair][0] = delta.x;
                            deltas[i_pair][1] = delta.y;
                            deltas[i_pair][2] = delta.z;
                            distances[i_pair] = distance;
                            if (requires_transpose) {
                                neighbors[0][i_pair + 1] = orj;
                                neighbors[1][i_pair + 1] = ori;
                                deltas[i_pair + 1][0] = -delta.x;
                                deltas[i_pair + 1][1] = -delta.y;
                                deltas[i_pair + 1][2] = -delta.z;
                                distances[i_pair + 1] = distance;
                            }
                        } // endif
                    }     // endif
                }         // endfor
            }             // endif
        }                 // endfor
    }                     // endfor
}

class Autograd : public Function<Autograd> {
public:
    static tensor_list forward(AutogradContext* ctx, const Tensor& positions, const Tensor& batch,
                               const Tensor& box_size, bool use_periodic,
                               const Scalar& cutoff_lower, const Scalar& cutoff_upper,
                               const Scalar& max_num_pairs, bool loop, bool include_transpose) {
        // The algorithm for the cell list construction can be summarized in three separate steps:
        //         1. Hash (label) the particles according to the cell (bin) they lie in.
        //         2. Sort the particles and hashes using the hashes as the ordering label
        //         (technically this is known as sorting by key). So that particles with positions
        //         lying in the same cell become contiguous in memory.
        //         3. Identify where each cell starts and ends in the sorted particle positions
        //         array.
        checkInput(positions, batch);
        TORCH_CHECK(box_size.dim() == 2, "Expected \"box_size\" to have two dimensions");
        TORCH_CHECK(box_size.size(0) == 3 && box_size.size(1) == 3,
                    "Expected \"box_size\" to have shape (3, 3)");
        TORCH_CHECK(box_size[0][1].item<double>() == 0 && box_size[0][2].item<double>() == 0 &&
                        box_size[1][0].item<double>() == 0 && box_size[1][2].item<double>() == 0 &&
                        box_size[2][0].item<double>() == 0 && box_size[2][1].item<double>() == 0,
                    "Expected \"box_size\" to be diagonal");
        const auto max_num_pairs_ = max_num_pairs.toInt();
        TORCH_CHECK(max_num_pairs_ > 0, "Expected \"max_num_neighbors\" to be positive");
        const int num_atoms = positions.size(0);
        const TensorOptions options = positions.options();
        // Steps 1 and 2
        Tensor sorted_positions, hash_values;
        std::tie(sorted_positions, hash_values) =
            sortPositionsByHash(positions, batch, box_size, cutoff_upper);
        Tensor cell_start, cell_end;
        // Step 3
        std::tie(cell_start, cell_end) =
            fillCellOffsets(sorted_positions, hash_values, batch, box_size, cutoff_upper);
        const Tensor neighbors = full({2, max_num_pairs_}, -1, options.dtype(kInt32));
        const Tensor deltas = empty({max_num_pairs_, 3}, options);
        const Tensor distances = full(max_num_pairs_, 0, options);
        const Tensor i_curr_pair = zeros(1, options.dtype(kInt32));
        const auto stream = getCurrentCUDAStream(positions.get_device());
        { // Traverse the cell list to find the neighbors
            const CUDAStreamGuard guard(stream);
            AT_DISPATCH_FLOATING_TYPES(positions.scalar_type(), "forward", [&] {
                const scalar_t cutoff_upper_ = cutoff_upper.to<scalar_t>();
                TORCH_CHECK(cutoff_upper_ > 0, "Expected cutoff_upper to be positive");
                const scalar_t cutoff_lower_ = cutoff_lower.to<scalar_t>();
                const scalar3<scalar_t> box_size_ = {box_size[0][0].item<scalar_t>(),
                                                     box_size[1][1].item<scalar_t>(),
                                                     box_size[2][2].item<scalar_t>()};
                const int threads = 128;
                const int blocks = (num_atoms + threads - 1) / threads;
                forward_kernel<<<blocks, threads, 0, stream>>>(
                    get_accessor<scalar_t, 2>(sorted_positions),
                    get_accessor<int32_t, 1>(hash_values), get_accessor<int64_t, 1>(batch),
                    get_accessor<int32_t, 1>(cell_start), get_accessor<int32_t, 1>(cell_end),
                    get_accessor<int32_t, 2>(neighbors), get_accessor<scalar_t, 2>(deltas),
                    get_accessor<scalar_t, 1>(distances), get_accessor<int32_t, 1>(i_curr_pair),
                    num_atoms, max_num_pairs_, use_periodic, box_size_, cutoff_lower_,
                    cutoff_upper_, loop, include_transpose);
            });
        }
        ctx->save_for_backward({neighbors, deltas, distances});
        ctx->saved_data["num_atoms"] = num_atoms;
        return {neighbors, deltas, distances, i_curr_pair};
    }

    static tensor_list backward(AutogradContext* ctx, tensor_list grad_inputs) {
        return common_backward(ctx, grad_inputs);
    }
};

TORCH_LIBRARY_IMPL(neighbors, AutogradCUDA, m) {
    m.impl("get_neighbor_pairs_cell",
           [](const Tensor& positions, const Tensor& batch, const Tensor& box_vectors,
              bool use_periodic, const Scalar& cutoff_lower, const Scalar& cutoff_upper,
              const Scalar& max_num_pairs, bool loop, bool include_transpose) {
               const tensor_list results =
                   Autograd::apply(positions, batch, box_vectors, use_periodic, cutoff_lower,
                                   cutoff_upper, max_num_pairs, loop, include_transpose);
               return std::make_tuple(results[0], results[1], results[2], results[3]);
           });
}
